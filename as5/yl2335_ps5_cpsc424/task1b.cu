#define FP double

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m) {

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int indexb = col;
  int index = row * m + col;
  
  if(col < m && row < n) {
    c[index] = 0.;
    for (int indexa = row * p; indexa < (row * p + p); indexa++, indexb += m) 
      c[index] += a[indexa] * b[indexb];
  }

}


void cpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m) {

  int index;
  for (int k = 0; k < p; k++)
    for (int row = 0; row < n; row++) {
      index = row * m;
      for (int col = 0; col < m; col++, index++) {
        c[index] -= a[row * p + k] * b[k * m + col];
      }
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_x = 1, Grid_Dim_y = 1; //Grid dimension, x and y, square
  int Block_Dim_x = 1, Block_Dim_y = 1; //Block dimension, x and y, square

  int n, m, p; // matrix dimensions
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  int sizeA, sizeB, sizeC; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if (argc != 5) {
    printf("Usage: matmul <matrix dim n> <matrix dim m> <matrix dim p> <block dim>\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);

  Block_Dim_x = Block_Dim_y = atoi(argv[4]); // Square block
  if (Block_Dim_x * Block_Dim_y > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim_x = (m - 1) / Block_Dim_x + 1;
  Grid_Dim_y = (n - 1) / Block_Dim_y + 1; 
  if (Grid_Dim_x * Block_Dim_x < m || Grid_Dim_y * Block_Dim_y < n) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit (-1);
  }

  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimensions = %d, %d, %d\n", n, p, m);
  printf("Block_Dim = (%d, %d), Grid_Dim = (%d, %d), \n", Block_Dim_x, Block_Dim_y, Grid_Dim_x, Grid_Dim_y);

  dim3 Grid(Grid_Dim_x, Grid_Dim_y); //Grid structure
  dim3 Block(Block_Dim_x, Block_Dim_y); //Block structure

  sizeA = n * p * sizeof(FP);
  sizeB = p * m * sizeof(FP);
  sizeC = n * m * sizeof(FP);

  a = (FP*) malloc(sizeA); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(sizeB);
  c = (FP*) malloc(sizeC); // results from GPU

  srand(12345);
  for(i = 0; i < n; i++)
    for(j = 0; j < p; j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i = 0; i < p; i++)
    for(j = 0; j < m; j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * m + j] = (FP) i+j; // may be helpful for debugging
    }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, sizeA); // allocate memory on device
  hipMalloc((void**)&dev_b, sizeB);
  hipMalloc((void**)&dev_c, sizeC);

  hipMemcpy(dev_a, a , sizeA ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , sizeB ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  gpu_matrixmult<<<Grid,Block>>>(dev_a, dev_b, dev_c, n, p, m);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);

  hipMemcpy(c, dev_c, sizeC, hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n\n", elapsed_time_ms); // exec. time

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
