#define FP float

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_matrixmult(FP *a, FP *b, FP *c, int n, int p, int m, int TW, int NTB) {

  extern __shared__ FP bigarray[];
  FP *atile = &bigarray[0], *btile = &bigarray[TW * TW];
  int tx = threadIdx.x, ty = threadIdx.y, bx = blockIdx.x, by = blockIdx.y;

  int baseRow = by * blockDim.y, baseCol = bx * NTB * TW;
  // if (baseRow >= n || baseCol >= m) return;

  int row = ty + baseRow, col = tx + baseCol;

  int indexa, indexb;

  // Loop over tiles
  for (int t = 0; t < (p - 1) / TW + 1; t++) {
    // Reset col
    col = tx + baseCol;

    indexa = row * p + t * TW + tx;
    atile[ty * TW + tx] = (row >= n || t * TW + tx >= p ? 0 : a[indexa]); // Copy to shared memory

    // Loop over muti-tiles
    for (int t1 = 0; t1 < NTB; t1++, col += TW) {
      if (col >= m) break;
      
      indexb = (t * TW + ty) * m + col;
      btile[ty * TW + tx] = (t * TW + ty >= p ? 0 : b[indexb]); // Copy to shared memory
      __syncthreads();

      // Compute tile ctile = atile x btile (each thread computes one element)
      if (row < n && col < m) 
        for (int k = 0; k < TW; k++)
          c[row * m + col] += atile[ty * TW + k] * btile[k * TW + tx];
      __syncthreads();
    }

  }

}


void cpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m) {

  int index;
  for (int k = 0; k < p; k++)
    for (int row = 0; row < n; row++) {
      index = row * m;
      for (int col = 0; col < m; col++, index++) {
        c[index] -= a[row * p + k] * b[k * m + col];
      }
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_x = 1, Grid_Dim_y = 1; //Grid dimension, x and y, square
  int Block_Dim_x = 1, Block_Dim_y = 1; //Block dimension, x and y, square
  int TW = 1;
  int NTB = 1;

  int n, m, p; // matrix dimensions
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  int sizeA, sizeB, sizeC; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if (argc != 6) {
    printf("Usage: matmul <matrix dim n> <matrix dim m> <matrix dim p> <block dim> <tiles>\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);
  NTB = atoi(argv[5]);

  TW = Block_Dim_x = Block_Dim_y = atoi(argv[4]); // Square block
  if (Block_Dim_x * Block_Dim_y > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  // Grid_Dim_x = (m - 1) / Block_Dim_x + 1;
  Grid_Dim_x = (m - 1) / (NTB * TW) + 1;
  Grid_Dim_y = (n - 1) / Block_Dim_y + 1; 
  if (Grid_Dim_x * NTB * TW < m || Grid_Dim_y * Block_Dim_y < n) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit (-1);
  }

  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimensions = %d, %d, %d\n", n, p, m);
  printf("Block_Dim = (%d, %d), Grid_Dim = (%d, %d), \n", Block_Dim_x, Block_Dim_y, Grid_Dim_x, Grid_Dim_y);

  dim3 Grid(Grid_Dim_x, Grid_Dim_y); //Grid structure
  dim3 Block(Block_Dim_x, Block_Dim_y); //Block structure

  sizeA = n * p * sizeof(FP);
  sizeB = p * m * sizeof(FP);
  sizeC = n * m * sizeof(FP);

  a = (FP*) malloc(sizeA); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(sizeB);
  c = (FP*) malloc(sizeC); // results from GPU

  srand(12345);
  for(i = 0; i < n; i++)
    for(j = 0; j < p; j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i = 0; i < p; i++)
    for(j = 0; j < m; j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * m + j] = (FP) i+j; // may be helpful for debugging
    }

  memset(c, 0., sizeC);

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, sizeA); // allocate memory on device
  hipMalloc((void**)&dev_b, sizeB);
  hipMalloc((void**)&dev_c, sizeC);

  hipMemcpy(dev_a, a , sizeA ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , sizeB ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  size_t Ns = 2 * TW * TW * sizeof(FP);
  gpu_matrixmult<<<Grid, Block, Ns>>>(dev_a, dev_b, dev_c, n, p, m, TW, NTB);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);

  hipMemcpy(c, dev_c, sizeC, hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed


  cpu_matrixmult(a, b, c, n, p, m); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for (i = 0; i < n * p; i++) {
    ai = (double) a[i];
    suma += ai * ai;
  }
  for (i = 0; i < p * m; i++) {
    bi = (double) b[i];
    sumb += bi * bi;
  }
  for (i = 0; i < n * m; i++) {
    ci = (double) c[i];
    sumc += ci * ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc / (sqrt(n * m) * suma * sumb);
  printf("Scaled error between GPU and CPU: %e\n\n", error);

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
