#define FP double

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m, int TW) {

  extern __shared__ FP bigarray[];
  FP *atile = &bigarray[0], *btile = &bigarray[TW * TW];
  int tx = threadIdx.x, ty = threadIdx.y, bx = blockIdx.x, by = blockIdx.y;
  int row = ty + by * blockDim.y, col = tx + bx * blockDim.x;
  FP cvalue = 0.;
  int indexa, indexb;

  // Loop over tiles
  for (int t = 0; t < (p - 1) / TW + 1; t++) {
    indexa = row * p + t * TW + tx;
    indexb = (t * TW + ty) * m + col;

    // Copy to shared memory
    atile[ty * TW + tx] = (row >= n || t * TW + tx >= p ? 0 : a[indexa]);
    // Copy to shared memory
    btile[ty * TW + tx] = (col >= m || t * TW + ty >= p ? 0 : b[indexb]);
    __syncthreads();

    // Compute tile ctile = atile x btile (each thread computes one element)
    for (int k = 0; k < TW; k++)
      cvalue += atile[ty * TW + k] * btile[k * TW + tx];
    __syncthreads();
  }

  if (row < n && col < m)
    c[row * m + col] = cvalue;
}


void cpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m) {

  int index;
  for (int k = 0; k < p; k++)
    for (int row = 0; row < n; row++) {
      index = row * m;
      for (int col = 0; col < m; col++, index++) {
        c[index] -= a[row * p + k] * b[k * m + col];
      }
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_x = 1, Grid_Dim_y = 1; //Grid dimension, x and y, square
  int Block_Dim_x = 1, Block_Dim_y = 1; //Block dimension, x and y, square
  int TW = 1; // Tile width

  int n, m, p; // matrix dimensions
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  int sizeA, sizeB, sizeC; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if (argc != 5) {
    printf("Usage: matmul <matrix dim n> <matrix dim m> <matrix dim p> <block dim>\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);

  TW = Block_Dim_x = Block_Dim_y = atoi(argv[4]); // Square block
  if (Block_Dim_x * Block_Dim_y > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim_x = (m - 1) / Block_Dim_x + 1;
  Grid_Dim_y = (n - 1) / Block_Dim_y + 1; 
  if (Grid_Dim_x * Block_Dim_x < m || Grid_Dim_y * Block_Dim_y < n) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit (-1);
  }

  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimensions = %d, %d, %d\n", n, p, m);
  printf("Block_Dim = (%d, %d), Grid_Dim = (%d, %d), \n", Block_Dim_x, Block_Dim_y, Grid_Dim_x, Grid_Dim_y);

  dim3 Grid(Grid_Dim_x, Grid_Dim_y); //Grid structure
  dim3 Block(Block_Dim_x, Block_Dim_y); //Block structure

  sizeA = n * p * sizeof(FP);
  sizeB = p * m * sizeof(FP);
  sizeC = n * m * sizeof(FP);

  a = (FP*) malloc(sizeA); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(sizeB);
  c = (FP*) malloc(sizeC); // results from GPU

  srand(12345);
  for(i = 0; i < n; i++)
    for(j = 0; j < p; j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i = 0; i < p; i++)
    for(j = 0; j < m; j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * m + j] = (FP) i+j; // may be helpful for debugging
    }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, sizeA); // allocate memory on device
  hipMalloc((void**)&dev_b, sizeB);
  hipMalloc((void**)&dev_c, sizeC);

  hipMemcpy(dev_a, a , sizeA ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , sizeB ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  size_t Ns = 2 * TW * TW * sizeof(FP);
  gpu_matrixmult<<<Grid, Block, Ns>>>(dev_a, dev_b, dev_c, n, p, m, TW);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);

  hipMemcpy(c, dev_c, sizeC, hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n\n", elapsed_time_ms); // exec. time

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
